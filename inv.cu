#include "hip/hip_runtime.h"
#include "lib/index.h"

using std::string;
using std::map;

int main(int argc, const char *argv[]){
	hipblasCreate(&device::cublas_handle);
	hipsolverDnCreate(&device::solver_handle);

	map<string, string> cfg;

	for (size_t i = 0; i < argc; i++) {
		string arg = argv[i];
		size_t pos = arg.find("=");
		if (pos != string::npos && arg[0] == '-') {
			string key = arg.substr(1, pos - 1);
			string value = arg.substr(pos + 1);
			cfg[key] = value;
		}
		else {
			pos = arg.find(".cfg");
			if (pos != string::npos) {
				cfg["config"] = arg;
			}
		}
	}

	if (!cfg["config"].size()) {
		std::cout << "Using example/checker.cfg" << std::endl;
		cfg["config"] = "example/checker.cfg";
	}

	Config *config = new Config(cfg);
	switch (config->i["mode"]) {
		case 0: {
			Solver *solver = module::solver(config->i["solver"]);
	        Filter *filter = module::filter(config->i["filter"]);
			Misfit *misfit = module::misfit(config->i["misfit"]);
			Optimizer *optimizer = module::optimizer(config->i["optimizer"]);

			misfit->init(config, solver, filter);
			optimizer->init(config, solver, misfit);
			optimizer->run();
			break;
		}
		case 1: {
			Solver *solver = module::solver(config->i["solver"]);
			solver->init(config);
			solver->importModel(true);
			solver->exportAxis();
			solver->runForward(-1, false, true, true);
			break;
		}
		case 2: {
			Solver *solver = module::solver(config->i["solver"]);
			Filter *filter = module::filter(config->i["filter"]);
			Misfit *misfit = module::misfit(config->i["misfit"]);

			misfit->init(config, solver, filter);
			misfit->calc(true);
			solver->exportAxis();
			solver->exportKernels();
			break;
		}
	}


	/* clock_t start = clock();
	solver->run(true);
	double duration = (clock() - start) / (double) CLOCKS_PER_SEC;
	cout << "Elapsed time: " << duration << endl; */

	hipblasDestroy(device::cublas_handle);
	hipsolverDnDestroy(device::solver_handle);

	return 0;
}
