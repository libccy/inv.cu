#include "hip/hip_runtime.h"
#include "lib/index.h"

using std::string;
using std::map;

int main(int argc, const char *argv[]){
	hipblasCreate(&device::cublas_handle);

	map<string, string> cfg;

	for (size_t i = 1; i < argc; i++) {
		string arg = argv[i];
		size_t pos = arg.find("=");
		if (pos != string::npos && arg[0] == '-') {
			string key = arg.substr(1, pos - 1);
			string value = arg.substr(pos + 1);
			cfg[key] = value;
		}
		else {
			cfg["config"] = arg;
		}
	}

	if (!cfg["config"].size()) {
		std::cout << "Using example/checker" << std::endl;
		cfg["config"] = "example/checker";
	}

	Config *config = new Config(cfg);
	switch (config->i["mode"]) {
		case 0: {
			Solver *solver = module::solver(config->i["solver"]);
	        Filter *filter = module::filter(config->i["filter"]);
			Misfit *misfit = module::misfit(config->i["misfit"]);
			Optimizer *optimizer = module::optimizer(config->i["optimizer"]);

			misfit->init(config, solver, filter);
			optimizer->init(config, solver, misfit);
			optimizer->run();
			break;
		}
		case 1: {
			Solver *solver = module::solver(config->i["solver"]);
			solver->init(config);
			solver->importModel(true);
			solver->exportAxis();
			solver->runForward(-1, false, true, true);
			break;
		}
		case 2: {
			Solver *solver = module::solver(config->i["solver"]);
			Filter *filter = module::filter(config->i["filter"]);
			Misfit *misfit = module::misfit(config->i["misfit"]);

			misfit->init(config, solver, filter);
			misfit->run(true);
			solver->exportAxis();
			solver->exportKernels();
			break;
		}
	}


	/* clock_t start = clock();
	solver->run(true);
	double duration = (clock() - start) / (double) CLOCKS_PER_SEC;
	cout << "Elapsed time: " << duration << endl; */

	hipblasDestroy(device::cublas_handle);

	return 0;
}
