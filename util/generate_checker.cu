#include "hip/hip_runtime.h"
#include "../lib/index.h"

namespace _generateChecker {
	__global__ void generate(
		size_t cx, size_t cz, size_t nx, size_t nz,
		float dx, float dz,
		float vp0, float vs0, float rho0,
		float dvp, float dvs, float drho,
		float *x, float *z,
		float *vp, float *vs, float *rho, Dim dim){
		size_t i, j, k;
		dim(i, j, k);
		x[k] = i * dx;
		z[k] = j * dz;
		float wx = dx * nx * 2 / 3 / (cx + 1);
		float wz = dz * nz * 2 / 3 / (cz + 1);
		size_t idx = (x[k] - wx) / wx / 1.5;
		size_t idz = (z[k] - wz) / wz / 1.5;

		float rx = x[k] - wx - idx * wx * 1.5;
		float rz = z[k] - wz - idz * wz * 1.5;

		if (rx > 0 && rx < wx && rz > 0 && rz < wz) {
			if (idx % 2 == idz % 2) {
				vp[k] = vp0 + dvp;
				vs[k] = vs0 + dvs;
				rho[k] = rho0 + drho;
			}
			else{
				vp[k] = vp0 - dvp;
				vs[k] = vs0 - dvs;
				rho[k] = rho0 - drho;
			}
		}
		else {
			vp[k] = vp0;
			vs[k] = vs0;
			rho[k] = rho0;
		}
	}
}

using std::map;
using std::string;
using namespace _generateChecker;

void generateChecker(
	size_t cx, size_t cz, size_t nx, size_t nz,
	float dx, float dz,
	float vp0, float vs0, float rho0,
	float dvp, float dvs, float drho, size_t sigma) {
	Dim dim(nx, nz);

	float *x = device::create(dim);
	float *z = device::create(dim);
	float *vp = device::create(dim);
	float *vs = device::create(dim);
	float *rho = device::create(dim);

	generate<<<dim.dg, dim.db>>>(
		cx, cz, nx, nz, dx, dz,
		vp0, vs0, rho0, dvp, dvs, drho,
		x, z, vp, vs, rho, dim
	);

	if (sigma) {
		Filter *filter = module::filter(0);
		filter->init(nx, nz, sigma);
		if (dvp > 0.1) filter->apply(vp);
		if (dvs > 0.1) filter->apply(vs);
		if (drho > 0.1) filter->apply(rho);
		delete filter;
	}

	int npt = nx * nz;
	float *buffer = host::create(npt);
	auto write = [&](string comp, float *data) {
		device::toHost(buffer, data, dim);
		std::ofstream outfile("output/proc000000_" + comp + ".bin", std::ofstream::binary);
		outfile.write(reinterpret_cast<char*>(&npt), sizeof(int));
		outfile.write(reinterpret_cast<char*>(buffer), npt * sizeof(float));
		outfile.close();
	};

	write("x", x);
	write("z", z);
	write("vp", vp);
	write("vs", vs);
	write("rho", rho);

	hipFree(x);
	hipFree(z);
	hipFree(vp);
	hipFree(vs);
	hipFree(rho);
	free(buffer);
}

int main(int argc, const char *argv[]){
	map<string, float> dict {
		{"cx", 4},
		{"cz", 4},
		{"nx", 201},
		{"nz", 201},
		{"sigma", 3},
		{"dx", 2400},
		{"dz", 2400},
		{"vp0", 0},
		{"vs0", 3500},
		{"rho0", 2600},
		{"dvp", 0},
		{"dvs", 400},
		{"drho", 0}
	};

	for (size_t i = 0; i < argc; i++) {
		string arg = argv[i];
		size_t pos = arg.find("=");
		if (pos != string::npos && arg[0] == '-') {
			string key = arg.substr(1, pos - 1);
			string value = arg.substr(pos + 1);

			std::istringstream f_valuestream(value);
			float f_value;
			f_valuestream >> f_value;
			if (f_valuestream.eof() && !f_valuestream.fail()) {
				dict[key] = f_value;
			}
		}
	}

	generateChecker(
		dict["cx"], dict["cz"], dict["nx"], dict["nz"],
		dict["dx"], dict["dz"], dict["vp0"], dict["vs0"], dict["rho0"],
		dict["dvp"], dict["dvs"], dict["drho"], dict["sigma"]
	);

	return 0;
}
